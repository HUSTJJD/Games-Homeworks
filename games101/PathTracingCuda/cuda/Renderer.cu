#include "hip/hip_runtime.h"
//
// Created by goksu on 2/25/20.
//
#include "../Renderer.hpp"
#include "../Scene.hpp"
#include "../BVH.hpp"

__device__ void BVHAccel::getSample(BVHBuildNode *node, float oPos, Intersection &pos, float &pdf, int thread_id)
{
    if (node->left == nullptr || node->right == nullptr)
    {

        if (node->object->type == EObjectType::MeshTriangle)
        {
            ((MeshTriangle *)node->object)->Sample(pos, pdf, thread_id);
        }
        else if (node->object->type == EObjectType::Triangle)
        {
            ((Triangle *)node->object)->Sample(pos, pdf, thread_id);
        }
        pdf *= node->area;
        return;
    }
    if (oPos < node->left->area)
        getSample(node->left, oPos, pos, pdf, thread_id);
    else
        getSample(node->right, oPos - node->left->area, pos, pdf, thread_id);
}

__device__ Intersection BVHAccel::getIntersection(BVHBuildNode *node, const Ray &ray) const
{
    // Home Work Begin
    Intersection inter;
    // 判断光线方向是否为负
    // 光线方向
    float x = ray.direction.x;
    float y = ray.direction.y;
    float z = ray.direction.z;
    // 判断坐标是否为负
    bool dirsIsNeg[3] = {x > 0, y > 0, z > 0};
    // 判断结点的包围盒与光线是否相交
    if (node->bounds.IntersectP(ray, ray.direction_inv, dirsIsNeg) == false)
        return inter;
    if (node->left == nullptr && node->right == nullptr)
    {
        if (node->object->type == EObjectType::MeshTriangle)
        {
            inter = ((MeshTriangle *)node->object)->getIntersection(ray);
        }
        else if (node->object->type == EObjectType::Triangle)
        {
            inter = ((Triangle *)node->object)->getIntersection(ray);
        }
        return inter;
    }
    // 递归判断子节点是否存在与光线相交的情况
    auto hit1 = getIntersection(node->left, ray);
    auto hit2 = getIntersection(node->right, ray);
    if (hit1.distance < hit2.distance)
        return hit1;
    return hit2;
}

__device__ Vector3f Scene::castRay(const Ray &ray, int depth, int thread_id) const
{
    Intersection inter = intersect(ray);
    if (inter.happened)
    {
        // 如果射线第一次打到光源，直接返回
        if (inter.m->hasEmission())
        {
            if (depth == 0)
                return inter.m->getEmission();
            else
                return Vector3f(0, 0, 0);
        }
        Vector3f L_dir(0, 0, 0);
        Vector3f L_indir(0, 0, 0);
        // 随机 sample 灯光，用该 sample 的结果判断射线是否击中光源
        Intersection lightInter;
        float pdf_light = 0.0f;
        sampleLight(lightInter, pdf_light, thread_id);
        // 物体表面法线
        auto &N = inter.normal;
        // 灯光表面法线
        auto &NN = lightInter.normal;
        auto &objPos = inter.coords;
        auto &lightPos = lightInter.coords;
        auto diff = lightPos - objPos;
        auto lightDir = diff.normalized();
        float lightDistance = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
        Ray light(&objPos, lightDir);
        Intersection light2obj = intersect(light);
        // 如果反射击中光源
        if (light2obj.happened && (light2obj.coords - lightPos).norm() < 1e-2)
        {
            Vector3f f_r = inter.m->eval(ray.direction, lightDir, N);
            L_dir = lightInter.emit * f_r * dotProduct(lightDir, N) * dotProduct(-lightDir, NN) / lightDistance / pdf_light;
        }
        if (cuda::get_random_float(thread_id) < RussianRoulette)
        {
            Vector3f nextDir = inter.m->sample(ray.direction, N, thread_id).normalized();
            Ray nextRay(&objPos, nextDir);
            Intersection nextInter = intersect(nextRay);
            if (nextInter.happened && !nextInter.m->hasEmission())
            {
                float pdf = inter.m->pdf(ray.direction, nextDir, N);
                Vector3f f_r = inter.m->eval(ray.direction, nextDir, N);
                L_indir = castRay(nextRay, depth + 1, thread_id) * f_r * dotProduct(nextDir, N) / pdf / RussianRoulette;
            }
        }
        return L_dir + L_indir;
    }
    return Vector3f(0, 0, 0);
}

__global__ void kernel(Scene *scene, int spp, Vector3f *eye_pos, float scale, float imageAspectRatio, Vector3f *framebuffer)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    int total_count = scene->width * scene->height * spp;
    while (tid < total_count)
    {
        // 修正索引计算：按像素和采样分解
        int pixel_id = tid / spp; // 像素ID
        int i = pixel_id % scene->width;
        int j = pixel_id / scene->width;

        // 检查i和j是否越界
        if (i >= scene->width || j >= scene->height)
        {
            tid += offset;
            printf("out\n");
            continue;
        }

        float x = (2 * (i + 0.5) / (float)scene->width - 1) * imageAspectRatio * scale;
        float y = (1 - 2 * (j + 0.5) / (float)scene->height) * scale;
        Vector3f dir = normalize(Vector3f(-x, y, 1));

        // 累加前确保原子操作或同步
        Vector3f color = scene->castRay(Ray(eye_pos, dir), 0, tid) / spp;
        atomicAdd(&framebuffer[j * scene->width + i].x, color.x);
        atomicAdd(&framebuffer[j * scene->width + i].y, color.y);
        atomicAdd(&framebuffer[j * scene->width + i].z, color.z);

        tid += offset;
    }
}

void validGPUAvailable()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0)
    {
        printf("no cuda gpu detected\n");
        exit(-1);
    }
    else
    {
        printf("cuda gpu available\n");
    }
}

void setCudaLimit()
{
    size_t size_heap, size_stack;
    // 当出现内存错误时，适当调整此处参数
    hipDeviceSetLimit(hipLimitMallocHeapSize, 10240000 * 8);
    hipDeviceSetLimit(hipLimitStackSize, 128 * 1024);
    hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
    hipDeviceGetLimit(&size_stack, hipLimitStackSize);
    printf("Heap size found to be %d; Stack size found to be %d\n", (int)size_heap, (int)size_stack);
}

// The main render function. This where we iterate over all pixels in the image,
// generate primary rays and cast these rays into the scene. The content of the
// framebuffer is saved to a file.
void Renderer::Render(Scene *scene, int spp, Vector3f &in_eye_pos, Vector3f *in_framebuffer)
{
    validGPUAvailable();
    setCudaLimit();

    auto pixel_num = scene->height * scene->width;

    Vector3f *framebuffer;
    hipMalloc((void **)&framebuffer, pixel_num * sizeof(Vector3f));
    hipMemcpy(framebuffer, in_framebuffer, pixel_num * sizeof(Vector3f), hipMemcpyHostToDevice);

    Vector3f *eye_pos;
    hipMalloc((void **)&eye_pos, sizeof(Vector3f));
    hipMemcpy(eye_pos, &in_eye_pos, sizeof(Vector3f), hipMemcpyHostToDevice);

    float scale = std::tan(scene->fov * 0.5 / 180.f * M_PI);
    float imageAspectRatio = (float)scene->width / (float)scene->height;

    int threads_per_block = spp;
    int blocks_needed = (pixel_num * spp + threads_per_block - 1) / threads_per_block;
    kernel<<<blocks_needed, threads_per_block>>>(scene, spp, eye_pos, scale, imageAspectRatio, framebuffer);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("After CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(in_framebuffer, framebuffer, pixel_num * sizeof(Vector3f), hipMemcpyDeviceToHost);
    hipFree(eye_pos);
    hipFree(framebuffer);
    printf("Render_GPU done\n");
}
